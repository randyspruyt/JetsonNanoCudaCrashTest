#include <hip/hip_runtime.h>

#include <unistd.h>
#include <future>
#include <mutex>
#include <stdio.h>

// This works fine with a mutex, but crashes with a sigbus error when not using a mutex
// #define USE_MUTEX

#ifdef USE_MUTEX
std::mutex m;
#endif

__global__ void testKernel() {
	printf("Thread Kernel running\n");
}

void testCuda() {
	testKernel<<<1,1>>>();
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("SYNC FAILED\n\n\n");
	}
}

struct MyThread {
	void run() {
		int threadLoop = 0;
		while(1) {
#ifdef USE_MUTEX
			m.lock();
#endif
			printf("Thread Run (loop %d)\n", threadLoop++);
			// run kernel
			testCuda();
#ifdef USE_MUTEX
			m.unlock();
#endif
			usleep(0);
		}
	}
};

int main(int argc, char** argv) {
	MyThread thread;
	auto threadFuture = std::async(std::launch::async, &MyThread::run, thread);
	int loop = 0;
	while(1){
#ifdef USE_MUTEX
		m.lock();
#endif
		int* tempHost = nullptr;
		int* tempDevice = nullptr;

		// 1.) Allocate mapped memory on the host
		printf("*** Main 1.) Allocating (loop = %d)\n", loop++);		
		hipError_t err = hipHostAlloc(&tempHost, sizeof(int), hipHostMallocMapped);
		if (err != hipSuccess) {
			printf("Failed to cudaHostAlloc()\n");
		}

		// get the device pointer (that is really mapped to the same memory as the host pointer)
		err = hipHostGetDevicePointer(&tempDevice, tempHost, 0);
		if (err != hipSuccess) {
			printf("Failed to cudaHostGetDevicePointer()\n");
		}

		// 2.) Set the host pointer to some value and read it
		*tempHost = 50;
		printf("*** Main 2.) Allocated mapped host value: %d\n", *tempHost);

		// 3.) If we copy the Device pointer back to a host value, we should get the same '50' value, 
		//	   since this is mapped memory
		int tempVal = 0;
		err = hipMemcpy(&tempVal, tempDevice, sizeof(int), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			printf("Failed to hipMemcpy () #1\n");
		}
		printf("*** Main 3.) Checking device value: %d\n", tempVal);

		// 4.) Copy new value of '89' from CPU to memory mapped device ptr
		tempVal = 89;
		err = hipMemcpy(tempDevice, &tempVal, sizeof(int), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			printf("Failed to cudaMemcpy() #2\n");
		}
		
		// reset tempVal
		tempVal = 0;
		
		// copy the device value back to tempVal
		err = hipMemcpy(&tempVal, tempDevice, sizeof(int), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			printf("Failed to cudaMemcpy() #3\n");
		}
		printf("*** Main 4.) Copy Back to Device: %d\n", tempVal);

		// 5.) Access tempHost to see if it is 89 as well, since tempDevice and tempHost are the same memory
		printf("*** Main 5.) Host Value: %d\n", *tempHost);

#ifdef USE_MUTEX
		m.unlock();
#endif
		usleep(0);
	}
}
