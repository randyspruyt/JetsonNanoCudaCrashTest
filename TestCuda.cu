#include <hip/hip_runtime.h>

#include <unistd.h>
#include <future>
#include <mutex>
#include <stdio.h>

// This works fine with a mutex, but crashes with a sigbus error when not using a mutex
// #define USE_MUTEX

#ifdef USE_MUTEX
std::mutex m;
#endif

__global__ void testKernel() {
	printf("Thread Kernel running\n");
}

void testCuda() {
	testKernel<<<1,1>>>();
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("SYNC FAILED\n\n\n");
	}
}

struct MyThread {
	void run() {
		int threadLoop = 0;
		while(1) {
#ifdef USE_MUTEX
			m.lock();
#endif
			printf("Thread Run (loop %d)\n", threadLoop++);
			// run kernel
			testCuda();
#ifdef USE_MUTEX
			m.unlock();
#endif
			usleep(0);
		}
	}
};

int main(int argc, char** argv) {
	MyThread thread;
	auto threadFuture = std::async(std::launch::async, &MyThread::run, thread);
	int loop = 0;
	while(1){
#ifdef USE_MUTEX
		m.lock();
#endif
		int* tempHost = nullptr;
		int* tempDevice = nullptr;
		printf("*** Main Allocating (loop = %d)\n", loop++);

		// allocate mapped memory on the host
		hipError_t err = hipHostAlloc(&tempHost, sizeof(int), hipHostMallocMapped);
		if (err != hipSuccess) {
			printf("Failed to cudaHostAlloc()\n");
		}

		// get the device pointer (that is really mapped to the same memory as the host pointer)
		err = hipHostGetDevicePointer(&tempDevice, tempHost, 0);
		if (err != hipSuccess) {
			printf("Failed to cudaHostGetDevicePointer()\n");
		}

		// set the host pointer to some value and read it
		*tempHost = 50;
		printf("*** Main Allocated mapped host value: %d\n", *tempHost);

		// the device pointer should have this same value
		int tempVal = 0;
		err = hipMemcpy(&tempVal, tempDevice, sizeof(int), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			printf("Failed to hipMemcpy () #1\n");
		}
		printf("*** Main Checking device value: %d\n", tempVal);

		// copy new value of '89' from CPU to memory mapped device ptr
		tempVal = 89;
		err = hipMemcpy(tempDevice, &tempVal, sizeof(int), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			printf("Failed to cudaMemcpy() #2\n");
		}
		
		// reset tempVal
		tempVal = 0;
		
		// copy device value back to tempVal
		err = hipMemcpy(&tempVal, tempDevice, sizeof(int), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			printf("Failed to cudaMemcpy() #3\n");
		}
		printf("*** Main Copy Back to Device: %d\n", tempVal);

		// access tempHost to see if it is 89 as well
		printf("*** Main Host Value: %d\n", *tempHost);

#ifdef USE_MUTEX
		m.unlock();
#endif
		usleep(0);
	}
}
